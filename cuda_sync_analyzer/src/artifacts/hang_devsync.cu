#include <hip/hip_runtime.h>


__global__ void 
VecAdd( int* A) {
	while(1) {
		A[0] = 0;
	}
}

int
main(int argc, char *argv[]) {
	int *d_A;
	hipMalloc((void**)&d_A, 4096);
	VecAdd<<<1, 1, 0>>>(d_A);
	hipDeviceSynchronize();
}
